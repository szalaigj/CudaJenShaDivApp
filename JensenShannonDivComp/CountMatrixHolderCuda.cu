#include "CountMatrixHolder.cuh"

typedef unsigned char BYTE;

extern const BYTE subSeqLength;

__constant__ BYTE d_subSeqLength;
__constant__ int d_sequenceLength;
__constant__ int d_subSeqNumber;

void CountMatrixHolderCuda::innerSetupCountArrays(std::string& sequence, int seqLen, int subSeqNumber, int countArraySize)
{
	// CUDA grid launch parameters
	int nThreads = 256;
	dim3 nT(nThreads);
	dim3 nB((seqLen + subSeqLength - 1) / subSeqLength);
	if (nB.x > 65535)
	{
		std::stringstream errmsg;
		errmsg << "ERROR: Block is too large:\n";
		errmsg << nB.x << " blocks. Max is 65535.\n";
		throw std::runtime_error(errmsg.str());
	}

	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_subSeqLength), &subSeqLength, sizeof(BYTE)));
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_sequenceLength), &seqLen, sizeof(int)));
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_subSeqNumber), &subSeqNumber, sizeof(int)));
	//cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(d_sequence), sequence.c_str(), (seqLen + 1) * sizeof(*(sequence.c_str()))));
	char * d_sequence;
	int sizeOfSeq = (seqLen + 1) * sizeof(char);
	hipMalloc((void **)&d_sequence, sizeOfSeq);
	char * seqChars = (char *)sequence.c_str();
	cudaCheckError(hipMemcpy(d_sequence, seqChars, sizeOfSeq, hipMemcpyHostToDevice));

	BYTE * d_countArrayA;
	BYTE * d_countArrayC;
	BYTE * d_countArrayG;
	BYTE * d_countArrayT;

	int size = countArraySize * sizeof(BYTE);
	hipMalloc((void **)&d_countArrayA, size);
	hipMalloc((void **)&d_countArrayC, size);
	hipMalloc((void **)&d_countArrayG, size);
	hipMalloc((void **)&d_countArrayT, size);

	setupCountArraysKernel << < nB, nT >> >(d_sequence, d_countArrayA, d_countArrayC, d_countArrayG, d_countArrayT);

	cudaCheckError(hipPeekAtLastError());
	cudaCheckError(hipDeviceSynchronize());

	cudaCheckError(hipMemcpy(countArrayA, d_countArrayA, size, hipMemcpyDeviceToHost));
	cudaCheckError(hipMemcpy(countArrayC, d_countArrayC, size, hipMemcpyDeviceToHost));
	cudaCheckError(hipMemcpy(countArrayG, d_countArrayG, size, hipMemcpyDeviceToHost));
	cudaCheckError(hipMemcpy(countArrayT, d_countArrayT, size, hipMemcpyDeviceToHost));

	hipFree(d_sequence);
	hipFree(d_countArrayA);
	hipFree(d_countArrayC);
	hipFree(d_countArrayG);
	hipFree(d_countArrayT);
}