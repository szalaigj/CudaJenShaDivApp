/*
* This implementation is based on the article:
*	Grosse, I., Bernaola-Galvan, P., Carpena, P., Roman-Roldan, R., Oliver, J., & Stanley, H. E. (2002).
*	Analysis of symbolic sequences using the Jensen-Shannon divergence.
*	Physical Review E, 65(4), 041905
*/

// standard library includes
#include <iostream>

// local CudaJenShaDivApp includes
#include "CudaBasicIncludes.cuh"
#include "../Utils/ShannonEntropyComputer.cpp"
#include "../Utils/FrequencyComputer.cpp"
#include "../Utils/JenShaDivComputer.cpp"
#include "../DataHandlingUtil/LoadData.cpp"
#include "../DataHandlingUtil/SubSeqWriter.cpp"
#include "../Utils/Splitter.cpp"
#include "../Utils/ChiSquaredCDFComputer.cpp"
#include "../Utils/UIncGamma.cpp"

//#include <gsl/gsl_sf_gamma.h>
//#include <gsl/gsl_randist.h>
//#include <gsl/gsl_cdf.h>

// used symbols in sequences
const std::string symbols = "ACGT";

// the following values are referred to in article (Table 1)
// where the number of symbols is 4
const float betaParam = 0.8f;
const float aParam = 2.44f;
const float bParam = -6.15f;
const float hUsedParams[3] = { betaParam, aParam, bParam };

const double significanceThreshold = 0.9;
const int minSeqLength = 13;

__constant__ int sequenceLength;
__constant__ double usedParams[3];

int main(int argc, char** argv)
{
	BaseDataLoader loader;
	ShannonEntropyComputer entropyComputer;
	FrequencyComputer frequencyComputer(symbols);
	JenShaDivComputer jenShaDivComputer(entropyComputer);
	UIncGamma uIncGamma;
	ChiSquaredCDFComputer chiSquaredCDFComputer(uIncGamma);
	Splitter splitter(frequencyComputer, jenShaDivComputer, chiSquaredCDFComputer);
	BaseSubSeqWriter subSeqWriter;
	std::string filename(argv[1]);
	std::string inputData = loader.loadData(filename);
	int len = inputData.length();
	//std::cout << inputData << std::endl;
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(sequenceLength), &len, sizeof(int)));
	cudaCheckError(hipMemcpyToSymbol(HIP_SYMBOL(usedParams), hUsedParams, 3 * sizeof(double)));
	//std::string seq = "ACAGCAGGATATGTCGTGCT";
	std::string seq = "ACACACACACACACGTGTGTGTGTGTG";
	double * frequencies = frequencyComputer.computeFrequency(&seq);
	double entropy = entropyComputer.computeEntropy(frequencies, symbols.length());
	std::cout << entropy << std::endl;
	std::string sequencePrefix = seq.substr(0, 5);
	std::string sequencePostfix = seq.substr(5);
	double * frequenciesPrefix = frequencyComputer.computeFrequency(&sequencePrefix);
	double * frequenciesPostfix = frequencyComputer.computeFrequency(&sequencePostfix);
	double weightPrefix = (double)sequencePrefix.length() / (double)seq.length();
	double weightPostfix = (double)sequencePostfix.length() / (double)seq.length();
	double divergence = jenShaDivComputer.computeDivergence(frequenciesPrefix, frequenciesPostfix,
		weightPrefix, weightPostfix, symbols.length(), symbols.length());
	std::cout << divergence << std::endl;
	std::vector<std::string> subsequences = splitter.split(inputData);
	std::string outputFilename(argv[2]);
	subSeqWriter.writeData(outputFilename, &subsequences);
	std::cout << "Press any character and press enter to continue..." << std::endl;
	char chr;
	std::cin >> chr;
}